#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

hipError_t jacobiCuda(const int cluserSize,float * oldRowNum,  float *x, const float *diagonal_values , const float *non_diagonal_values, const int *indeces,const float *y, const int size);

__global__ void jacobiOne(const int clusterSize,float *oldRowNum,float *x, const float *diagonal_values , const float *non_diagonal_values, const int *indeces ,const float *y, const int size)
{
    const int index = blockDim.x * blockIdx.x + threadIdx.x; 
	float sum = 0 ;
	int oldIdx; // the old row index of the row being processed in thread
	

	if (index < size)
	{
		oldIdx = oldRowNum[index];
		for (int j = 0 ; j< 30 ; j++)
		{
			for (int i = 0 ; i<2 ; i++)
			{
				sum += non_diagonal_values[ i +2 * index]  * x[indeces[i  +2 * index]] ;
			}
			
			x[oldIdx] = (y[oldIdx] - sum )/diagonal_values[oldIdx];
			sum = 0 ;
			__syncthreads();	
		}
	}
}


__global__ void jacobiOneSharedAndLocal(const int clusterSize,float* oldRowNum,float *x, const float *diagonal_values , const float *non_diagonal_values, const int *indeces ,const float *y, const int size)
{
    const int index = blockDim.x * blockIdx.x + threadIdx.x;
	float local_diagonal_value ;
	float local_non_diagonal_values[2];
	float local_y;
	int oldIdx ;
	
	__shared__ float shared_x[4]; // shared_x[clusterSize + 2] ...... lma yossef yshof el dynamic allocation.
	

	local_diagonal_value = diagonal_values[index];
	local_non_diagonal_values[0] = non_diagonal_values[2 * index];
	local_non_diagonal_values[1] = non_diagonal_values[2 * index+1];
	local_y = y[index];
	shared_x[threadIdx.x + 1] = 0; // initialize the shared memory location as 0
	shared_x[0] = 0;//fill first and last positions with dummy values
	shared_x[3]=0;

	float sum = 0 ;
	if (threadIdx.x < clusterSize) // ensure you are withing the cluster
	{
		oldIdx = oldRowNum[index];
		for (int j = 0 ; j< 30 ; j++)
		{
			for (int i = 0 ; i<2 ; i++)
			{
				sum += local_non_diagonal_values[i]  * shared_x[i*2 + threadIdx.x ] ; //all memory access is in shared memory
			}
			shared_x[threadIdx.x+1] = (local_y - sum )/local_diagonal_value;
			sum = 0 ;
			__syncthreads();	
		}
		x[oldIdx] = shared_x[threadIdx.x+1];
		
	}
	
}



int main()
{
	//initialize our test cases
    const int arraySize = 12;
	const int clusterSize =2;
	float oldRowNum[12] = {0,1,2,7,3,8,4,6,5,10,9,11};
	float non_diagonal_values[24] = {0,0.0185,0.0185,0,0,0.0185,0.0185,0,0,0.0185,0.0185,0,0,0.0185,0.0185,0,0,0.0185,0.0185,0,0,0.0185,0.0185,0} ;
	float diagonal_values[12] = {};
	int indeces[24] = {0,1,0,0,0,7,2,0,0,8,3,0,0,6,4,0,0,10,5,0,0,11,9,0};
    float x[arraySize] = { 0};
	float y[arraySize] = {};
	for (int i = 0 ; i<arraySize ; i++)
	{
		y[i] = 0.0878 ;
		diagonal_values[i] = 0.0741;
	}

	
    hipError_t cudaStatus = jacobiCuda(clusterSize,oldRowNum,x,diagonal_values, non_diagonal_values, indeces,y, arraySize);
    if (cudaStatus != hipSuccess) {
        printf("%c\n", "JacobiCuda failed!");
        return 1;
    }

	printf("%f\n",x[0]);
	printf("%f\n",x[1]);
	printf("%f\n",x[2]);

	// hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        printf("%c\n", "hipDeviceReset failed!");
        return 1;
    }
	system("pause");
    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t jacobiCuda(const int clusterSize,float *oldRowNum, float *x, const float *diagonal_values , const float *non_diagonal_values, const int *indeces,const float *y, const int size)
{
    float *dev_non_diagonal_values = 0;
	float *dev_diagonal_values = 0;
    int *dev_indeces = 0;
	float *dev_y = 0 ;
    float *dev_x = 0;
	float *dev_old = 0;
	


	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);

    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
	

    if (cudaStatus != hipSuccess) {
		printf("%c\n", "hipSetDevice failed!");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_x, size * sizeof(float));
    if (cudaStatus != hipSuccess) {
        printf("%c\n", "hipMalloc for x failed!");
        goto Error;
    }

	cudaStatus = hipMalloc((void**)&dev_diagonal_values, size * sizeof(float));
    if (cudaStatus != hipSuccess) {
        printf("%c\n", "hipMalloc for diagonal values failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_non_diagonal_values, 2 * size * sizeof(float));
    if (cudaStatus != hipSuccess) {
        printf("%c\n", "hipMalloc for non diagonal values failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_indeces,  2 * size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        printf("%c\n", "hipMalloc for indeces failed!");
        goto Error;
    }

	cudaStatus = hipMalloc((void**)&dev_y, size * sizeof(float));
    if (cudaStatus != hipSuccess) {
        printf("%c\n", "hipMalloc for y failed!");
        goto Error;
    }
	
	cudaStatus = hipMalloc((void**)&dev_old,   size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        printf("%c\n", "hipMalloc for old num failed!");
        goto Error;
    }
    // Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_diagonal_values, diagonal_values, size * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        printf("%c\n", "hipMemcpy for diagonal values failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_non_diagonal_values, non_diagonal_values, 2 * size * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        printf("%c\n", "hipMemcpy for non_diagonal values failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_indeces, indeces, 2 * size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        printf("%c\n", "hipMemcpy for indeces failed!");
        goto Error;
    }

	cudaStatus = hipMemcpy(dev_y, y, size * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        printf("%c\n", "hipMemcpy for y failed!");
        goto Error;
    }

	cudaStatus = hipMemcpy(dev_x, x, size * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        printf("%c\n", "hipMemcpy for x failed!");
        goto Error;
    }
	
	cudaStatus = hipMemcpy(dev_old, oldRowNum, size * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        printf("%c\n", "hipMemcpy for old num values failed!");
        goto Error;
    }
    // Launch a kernel on the GPU with one thread for each element.
	hipEventRecord(start, 0);
	
	const dim3 blockDim(clusterSize,1,1);
	const dim3 gridDim(6,1,1);
    jacobiOneSharedAndLocal<<<gridDim, blockDim>>>(clusterSize,dev_old,dev_x, dev_diagonal_values , dev_non_diagonal_values , dev_indeces , dev_y , size);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	printf ("Time for the kernel: %f ms\n", time);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
		printf("%c\n", "JacobiOne launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        printf( "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(x, dev_x, size * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        printf("%c\n", "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_x);
    hipFree(dev_non_diagonal_values);
    hipFree(dev_indeces);
	hipFree(dev_old);
	/*hipFree(dev_numCols);
	hipFree(dev_numDiags);*/
    return cudaStatus;
}
