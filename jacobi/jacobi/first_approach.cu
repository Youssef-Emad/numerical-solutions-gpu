#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

void jacobiFirst();

__global__ void jacobiOne(float *x, const float *diagonal_values , const float *non_diagonal_values, const int *indeces ,const float *y, const int size)
{
    const int index = threadIdx.x;
	float sum = 0 ;

	if (index < size)
	{
		for (int j = 0 ; j< 30 ; j++)
		{
			for (int i = 0 ; i<2 ; i++)
			{
				sum += non_diagonal_values[2*index + i]  * x[indeces[2*index + i]] ;
			}
			x[index] = (y[index] - sum )/diagonal_values[index];
			sum = 0 ;
			__syncthreads();	
		}
	}
}

__global__ void jacobiOneShared(float *x, const float *diagonal_values , const float *non_diagonal_values, const int *indeces ,const float *y, const int size)
{
    const int index = threadIdx.x;
	__shared__ float shared_diagonal_values[24] ;
	__shared__ float shared_non_diagonal_values[48];
	__shared__ int shared_indeces[48];
	__shared__ float shared_y[24];
	__shared__ float shared_x[24];

	shared_diagonal_values[index] = diagonal_values[index];
	shared_non_diagonal_values[2*index] = non_diagonal_values[2*index];
	shared_non_diagonal_values[2*index+1] = non_diagonal_values[2*index+1];
	shared_indeces[2*index] = indeces[2*index];
	shared_indeces[2*index+1] = indeces[2*index+1];
	shared_y[index] = y[index];
	shared_x[index] = x[index];

	float sum = 0 ;
	if (index < size)
	{
		for (int j = 0 ; j< 30 ; j++)
		{
			for (int i = 0 ; i<2 ; i++)
			{
				sum += shared_non_diagonal_values[2*index + i]  * shared_x[shared_indeces[2*index + i]] ;
			}
			shared_x[index] = (shared_y[index] - sum )/shared_diagonal_values[index];
			sum = 0 ;
			__syncthreads();	
		}
		x[index] = shared_x[index];
	}
}

__global__ void jacobiOneSharedAndLocal(float *x, const float *diagonal_values , const float *non_diagonal_values, const int *indeces ,const float *y, const int size)
{
    const int index = threadIdx.x;
	float local_diagonal_value ;
	float local_non_diagonal_values[2];
	int local_indeces[2];
	float local_y;
	__shared__ float shared_x[24];

	local_diagonal_value = diagonal_values[index];
	local_non_diagonal_values[0] = non_diagonal_values[2*index];
	local_non_diagonal_values[1] = non_diagonal_values[2*index+1];
	local_indeces[0] = indeces[2*index];
	local_indeces[1] = indeces[2*index+1];
	local_y = y[index];
	shared_x[index] = x[index];

	float sum = 0 ;
	if (index < size)
	{
		for (int j = 0 ; j< 30 ; j++)
		{
			for (int i = 0 ; i<2 ; i++)
			{
				sum += local_non_diagonal_values[i]  * shared_x[local_indeces[i]] ;
			}
			
			shared_x[index] = (local_y - sum )/local_diagonal_value;
			sum = 0 ;
			__syncthreads();	
		}
		x[index] = shared_x[index];
	}
}

void jacobiFirst()
{
	//initialize our test cases
    const int arraySize = 24;
	/*float non_diagonal_values[] ={3,2,1,2,2,1};
	float diagonal_values[3] ={5,6,7};
	int indeces[] ={1,2,0,2,0,1};
	int y[arraySize]= {14,13,24};*/

	/*float non_diagonal_values[] = {0.0185,0,0.0185,0,0.0185,0,0.0185,0,0.0185,0,0.0185,0,0.0185,0,0.0185,0,0.0185,0,0.0185,0,0.0185,0,0.0185,0} ;
	float diagonal_values[12] = {};
	int indeces[2*arraySize] = {0};
    float x[arraySize] = { 0 };
	float y[arraySize] = {};
	for (int i = 0 ; i<12 ; i++)
	{
		y[i] = 0.0878 ;
		diagonal_values[i] = 0.0741;
	}*/

	float non_diagonal_values[] = {0.0104,0, 0.0104, 0.0104, 0.0104,0, 0.0104,0, 0.0104,0, 0.0104,0, 0.0104,0, 0.0104, 0.0104, 0.0104,0, 0.0104,0, 0.0104, 0.0104, 0.0104, 0.0104, 0.0104,0, 0.0104, 0.0104, 0.0104, 0.0104, 0.0104, 0.0104, 0.0104,0, 0.0104,0, 0.0104,0, 0.0104,0, 0.0104,0, 0.0104, 0.0104, 0.0104,0, 0.0104,0} ;
	float diagonal_values[24] = {};
	int indeces[2*arraySize] = {1,1,0,2,1,1,10,10,11,11,7,7,13,13,5,9,15,15,7,7,3,17,4,18,14,14,6,20,12,16,8,22,14,14,10,10,11,11,21,21,13,13,19,23,15,15,21,21};
    float x[arraySize] = { 0 };
	float y[arraySize] = {0.0420,0.0594,0.0420,0.0420,0.0420,0.0420,0.0420,0.0594,0.0420,0.0420, 0.0594, 0.0594, 0.0420,0.0594,0.0594,0.0594, 0.0420, 0.0420, 0.0420, 0.0420, 0.0420,0.0594,0.0420,0.0420};
	for (int i = 0 ; i<24 ; i++)
	{
		diagonal_values[i] =  0.0417;
	}

	/*float non_diagonal_values[8] = {0} ;
	float diagonal_values[] = {0.1667,0.1667,0.1667,0.1667} ;
	int indeces[8] = {0};
	float y[] = {0.2036,0.2036,0.2036,0.2036};
	float x[arraySize] = { 0 };*/

    float *dev_non_diagonal_values = 0;
	float *dev_diagonal_values = 0;
    int *dev_indeces = 0;
	float *dev_y = 0 ;
    float *dev_x = 0;

    hipSetDevice(0);
	

    // Allocate GPU buffers
    hipMalloc((void**)&dev_x, size * sizeof(float));
    hipMalloc((void**)&dev_non_diagonal_values, 2 * size * sizeof(float));
    hipMalloc((void**)&dev_indeces, 2 * size * sizeof(int));
	hipMalloc((void**)&dev_y, size * sizeof(float));
   
    // Copy input vectors from host memory to GPU buffers.
	hipMemcpyAsync(dev_diagonal_values, diagonal_values, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpyAsync(dev_non_diagonal_values, non_diagonal_values, 2 * size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpyAsync(dev_indeces, indeces, 2 * size * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpyAsync(dev_y, y, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpyAsync(dev_x, x, size * sizeof(float), hipMemcpyHostToDevice);
    
    // Launch a kernel on the GPU with one thread for each element.
    jacobiOneSharedAndLocal<<<1, size>>>(dev_x, dev_diagonal_values , dev_non_diagonal_values , dev_indeces , dev_y , size);

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
	 hipDeviceSynchronize();
    // Copy output vector from GPU buffer to host memory.
    hipMemcpy(x, dev_x, size * sizeof(float), hipMemcpyDeviceToHost);
    
Error:
    hipFree(dev_x);
	hipFree(dev_y);
    hipFree(dev_diagonal_values);
    hipFree(dev_non_diagonal_values);
    hipFree(dev_indeces);
	hipDeviceReset();
}