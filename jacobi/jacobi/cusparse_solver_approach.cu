#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <hipsparse.h>

void cusparse_solver()
{
	//initialize our test cases
    const int m = 4;
	const int nnz = 4 ;
	const float alpha = 1.0;

	float values[] = {0,0,0,0} ;
	float diagonal_values[] = {0.1667,0.1667,0.1667,0.1667} ;
	int rowPtr[] = {0,1,2,3,4,7};
	int colIdx[] = {0,1,2,3};

	float y[] = {0.2036,0.2036,0.2036,0.2036};
	float x[4] ;

	float *dev_values = 0 ;
	float *dev_diagonal_values = 0 ;
	int *dev_rowPtr = 0 ;
	int *dev_colIdx = 0 ;
	float *dev_x = 0 ;
	float *dev_y = 0 ;
 
	//Define the cusparse opaque structures
    hipsparseHandle_t handle;
    hipsparseCreate(&handle);
	cusparseSolveAnalysisInfo_t info = 0;
	cusparseCreateSolveAnalysisInfo(&info);
	hipsparseMatDescr_t descr = 0;

    hipsparseCreateMatDescr(&descr);
    hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);

    // Choose which GPU to run on, change this on a multi-GPU system.
    hipSetDevice(0);

	// Allocate GPU buffers for three vectors (two input, one output)    .
    hipMalloc((void**)&dev_x, m * sizeof(float));
	hipMalloc((void**)&dev_y, m * sizeof(float));
	hipMalloc((void**)&dev_values, nnz * sizeof(float));
	hipMalloc((void**)&dev_diagonal_values, nnz * sizeof(float));
	hipMalloc((void**)&dev_rowPtr, (m+1) * sizeof(int));
	hipMalloc((void**)&dev_colIdx, nnz * sizeof(int));
  
	//Memcpy
	hipMemcpyAsync(dev_x, x, m * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpyAsync(dev_values, values, nnz * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpyAsync(dev_diagonal_values, diagonal_values, nnz * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpyAsync(dev_rowPtr, rowPtr, (m+1) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpyAsync(dev_colIdx, colIdx, nnz * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpyAsync(dev_y, y, m * sizeof(float), hipMemcpyHostToDevice);

	cusparseScsrsv_analysis(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, m, descr, dev_diagonal_values, dev_rowPtr, dev_colIdx, info);
	cusparseScsrsv_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, &alpha, descr, dev_diagonal_values, dev_rowPtr, dev_colIdx, info, dev_y, dev_x);
	
	hipMemcpyAsync(x, dev_x, m*sizeof(float), hipMemcpyDeviceToHost );


	// hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    hipDeviceReset();
   	cusparseDestroySolveAnalysisInfo(info);
	hipsparseDestroy(handle);
    hipFree(dev_x);
    hipFree(dev_y);
    hipFree(dev_values);
	hipFree(dev_rowPtr);
	hipFree(dev_colIdx);
}

