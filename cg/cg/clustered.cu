#include "hip/hip_runtime.h"

#include <stdio.h>
#include<stdlib.h>
#include <string.h>
#include <math.h>

char* concat(char *s1, char *s2);

__global__ void cg_variable_start(float* a , float* x,float * b ,int size) 
{
	int index = blockDim.x * blockIdx.x + threadIdx.x ;
	int local_index = threadIdx.x ;
	int block_index = blockIdx.x ;

	__shared__ float shared_r_squared[1024] ;
	__shared__ float shared_p_sum[1024] ;
	__shared__ float r[1024] ;
	float local_r ;

	shared_r_squared[local_index] = 0 ;
	shared_p_sum[local_index] = 0;
	__syncthreads() ;
	
	if (index < size)
	{
		float sum = 0 ;
		
		for (int i = 0 ; i<3 ; i++)
		{
			sum += a[3 * index  + i] * x[3 * block_index + i] ;
		}
		
		local_r = b[index] - sum ;	
		r[local_index] = local_r ;
		__syncthreads() ;

		for (int i = 0 ; i<3 ; i++)
		{
			shared_p_sum[local_index] += a[3*index  + i] * r[3* block_index + i] ;
		}
		__syncthreads() ;

		shared_r_squared[local_index] = local_r * local_r ;
		shared_p_sum[local_index] = shared_p_sum[local_index] * local_r ;
	}
	
	__syncthreads() ;

	for (unsigned int s = blockDim.x/2 ; s> 0 ; s >>= 1)
	{	
		if (threadIdx.x < s)
		{
			shared_r_squared[local_index] = shared_r_squared[local_index] + shared_r_squared[local_index +s] ;
			shared_p_sum[local_index] = shared_p_sum[local_index] + shared_p_sum[local_index +s] ;
			__syncthreads() ;
		}
			
	}	
	__syncthreads();

	if (index < size)
	{
		float alpha = shared_r_squared[0]/shared_p_sum[0] ;
		x[index] = x[index] + alpha * local_r ;	
	}

}

__global__ void cg_zero_start(float* a , float* x,float * b ,int size) 
{
	int index = blockDim.x * blockIdx.x + threadIdx.x ;
	int local_index = threadIdx.x ;
	int block_index = blockIdx.x ;
	
	__shared__ float shared_r_squared[1024] ;
	__shared__ float shared_p_sum[1024] ;
	float local_b ;
	
	shared_r_squared[local_index] = 0 ;
	shared_p_sum[local_index] = 0;
	__syncthreads() ;
	
	if (index < size)
	{
		local_b = b[index] ;

		for (int i = 0 ; i<3 ; i++)
		{
			shared_p_sum[local_index] += a[3*index  + i] * b[3* block_index + i] ;
		}
		__syncthreads() ;

		shared_r_squared[local_index] = local_b * local_b ;
		shared_p_sum[local_index] = shared_p_sum[local_index] * local_b ;
	}
	
	__syncthreads() ;

	for (unsigned int s = blockDim.x/2 ; s> 0 ; s >>= 1)
	{	
		if (threadIdx.x < s)
		{
			shared_r_squared[local_index] = shared_r_squared[local_index] + shared_r_squared[local_index +s] ;
			shared_p_sum[local_index] = shared_p_sum[local_index] + shared_p_sum[local_index +s] ;
			__syncthreads() ;
		}
			
	}	
	__syncthreads();

	if (index < size)
	{
		float alpha = shared_r_squared[0]/shared_p_sum[0] ;
		x[index] = x[index] + alpha * local_b ;	
	}

}
