#include "hip/hip_runtime.h"

#include <stdio.h>
#include<stdlib.h>
#include <string.h>
#include <math.h>

char* concat(char *s1, char *s2);

__global__ void cg_variable_start(float* a , float* x,float * b ,int size) 
{
	int index = blockDim.x * blockIdx.x + threadIdx.x ;
	int local_index = threadIdx.x ;
	int block_index = blockIdx.x ;

	__shared__ float shared_r_squared[1024] ;
	__shared__ float shared_p_sum[1024] ;
	__shared__ float r[1024] ;
	float local_r ;

	shared_r_squared[local_index] = 0 ;
	shared_p_sum[local_index] = 0;
	__syncthreads() ;
	
	if (index < size)
	{
		float sum = 0 ;
		
		for (int i = 0 ; i<3 ; i++)
		{
			sum += a[3 * index  + i] * x[3 * block_index + i] ;
		}
		
		local_r = b[index] - sum ;	
		r[local_index] = local_r ;
		__syncthreads() ;

		for (int i = 0 ; i<3 ; i++)
		{
			shared_p_sum[local_index] += a[3*index  + i] * r[3* block_index + i] ;
		}
		__syncthreads() ;

		shared_r_squared[local_index] = local_r * local_r ;
		shared_p_sum[local_index] = shared_p_sum[local_index] * local_r ;
	}
	
	__syncthreads() ;

	for (unsigned int s = blockDim.x/2 ; s> 0 ; s >>= 1)
	{	
		if (threadIdx.x < s)
		{
			shared_r_squared[local_index] = shared_r_squared[local_index] + shared_r_squared[local_index +s] ;
			shared_p_sum[local_index] = shared_p_sum[local_index] + shared_p_sum[local_index +s] ;
			__syncthreads() ;
		}
			
	}	
	__syncthreads();

	if (index < size)
	{
		float alpha = shared_r_squared[0]/shared_p_sum[0] ;
		x[index] = x[index] + alpha * local_r ;	
	}

}

__global__ void cg_zero_start(float* a , float* x,float * b ,int size) 
{
	int index = blockDim.x * blockIdx.x + threadIdx.x ;
	int local_index = threadIdx.x ;
	int block_index = blockIdx.x ;
	
	__shared__ float shared_r_squared[1024] ;
	__shared__ float shared_p_sum[1024] ;
	float local_b ;
	
	shared_r_squared[local_index] = 0 ;
	shared_p_sum[local_index] = 0;
	__syncthreads() ;
	
	if (index < size)
	{
		local_b = b[index] ;

		for (int i = 0 ; i<3 ; i++)
		{
			shared_p_sum[local_index] += a[3*index  + i] * b[3* block_index + i] ;
		}
		__syncthreads() ;

		shared_r_squared[local_index] = local_b * local_b ;
		shared_p_sum[local_index] = shared_p_sum[local_index] * local_b ;
	}
	
	__syncthreads() ;

	for (unsigned int s = blockDim.x/2 ; s> 0 ; s >>= 1)
	{	
		if (threadIdx.x < s)
		{
			shared_r_squared[local_index] = shared_r_squared[local_index] + shared_r_squared[local_index +s] ;
			shared_p_sum[local_index] = shared_p_sum[local_index] + shared_p_sum[local_index +s] ;
			__syncthreads() ;
		}
			
	}	
	__syncthreads();

	if (index < size)
	{
		float alpha = shared_r_squared[0]/shared_p_sum[0] ;
		x[index] = x[index] + alpha * local_b ;	
	}

}

void cg_clustered(const int size , char* file_name)
{
	//initialize our test cases

	float *values = (float *)malloc(3 * size * sizeof(float));
	float *x = (float *)malloc(size * sizeof(float));
	float *y = (float *)malloc(size * sizeof(float));
	float *output = (float *)malloc(size * sizeof(float));
	

	char* values_file_name = concat(file_name,"/basic/values.txt") ;
	char* y_file_name = concat(file_name,"/right_hand_side.txt");
	char* output_file_name = concat(file_name,"/output.txt");

	FILE *values_file = fopen(values_file_name, "r");
	FILE *y_file = fopen(y_file_name, "r");
	FILE *output_file = fopen(output_file_name, "r");

	for (int i = 0 ; i < size ; i++)
	{	
		fscanf(y_file, "%f", &y[i]);
		fscanf(output_file, "%f", &output[i]);
		x[i] = 0 ;
	}

	for (int i = 0 ; i< 3 * size ; i++)
	{
		fscanf(values_file, "%f", &values[i]);
	}
	
	float* dev_values = 0;
	float* dev_y = 0;
	float* dev_x = 0;
	
    hipSetDevice(0);
	
    // Allocate GPU buffers
    hipMalloc((void**)&dev_values, 3 * size * sizeof(float));
    hipMalloc((void**)&dev_y, size * sizeof(float));
    hipMalloc((void**)&dev_x, size * sizeof(float));
	
   
    // Copy input vectors from host memory to GPU buffers.
	hipMemcpyAsync(dev_values, values, 3 * size * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpyAsync(dev_y, y, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpyAsync(dev_x, x, size * sizeof(float), hipMemcpyHostToDevice);
	
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

    // Launch a kernel on the GPU with one thread for each row.
	cg_zero_start<<<350,698>>>(dev_values , dev_x,dev_y ,size) ;
	// cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
	hipDeviceSynchronize();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	printf ("Time for the kernel: %f ms\n", time);

    // Copy output vector from GPU buffer to host memory.
    hipMemcpy(x, dev_x, size * sizeof(float), hipMemcpyDeviceToHost);

	printf("%f\n",x[0]);
	printf("%f\n",x[1]);
	printf("%f\n",x[2]);
	printf("%f\n",x[size -2]);
	printf("%f\n",x[size -1]);
	
	hipFree(dev_values);
	hipFree(dev_y);
	hipFree(dev_x);
	
	hipDeviceReset();
	system("pause");
}

int main()
{
	cg_clustered(244300,"C:/Users/youssef/Desktop/numerical-solutions-gpu/cg/cg/test_cases/244300");
	return 1 ;
}