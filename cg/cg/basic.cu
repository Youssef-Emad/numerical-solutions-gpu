#include "hip/hip_runtime.h"

#include <stdio.h>
#include<stdlib.h>
#include <string.h>
#include <math.h>

char* concat(char *s1, char *s2);


__global__ void cg_full_global(float* a , int * indeces , float* b , float* x,float * r ,float * r_squared ,float * p_sum ,int size) 
{
	int index = blockDim.x * blockIdx.x + threadIdx.x ;
	/*float local_a[3] = {a[3*index],a[3*index + 1],a[3*index + 2]} ;
	int local_indeces[3]  = {indeces[3*index],indeces[3*index + 1],indeces[3*index + 2]} ;*/

	if (index < size)
	{
		float sum = 0 ;
		
		for (int i = 0 ; i<3 ; i++)
		{
			sum += a[3*index  + i] * x[indeces[3*index + i]] ;
		}
		
		float local_r = b[index] - sum ;	
		r[index] = local_r;

		p_sum[index] = 0 ;
		__syncthreads();

		for (int i = 0 ; i<3 ; i++)
		{
			p_sum[index] += a[3*index  + i] * r[indeces[3*index + i]] ;
		}
		
		//calc alpha
		r_squared[index] = local_r * local_r ;
		p_sum[index] = p_sum[index] * local_r ;
		
		//sum inside block
		for (unsigned int s = blockDim.x/2 ; s> 0 ; s >>= 1)
		{	
			if (threadIdx.x < s/2)
			{
				// summation of r*rT
				r_squared[index] = r_squared[index] + r_squared[index + s] ;
				//summation of r*a*rT
				p_sum[index] = p_sum[index] +  p_sum[index + s] ;
			}
			__syncthreads();
		}
		
		// sum between blocks - 1024 or 2048 blocks only
		int max_index_of_needed_blocks = gridDim.x/1025 ;

		if (blockIdx.x <= max_index_of_needed_blocks && gridDim.x > 1)
		{
			if(threadIdx.x == 0)
			{
				r_squared[blockIdx.x] = r_squared[blockIdx.x * blockDim.x] ;
				p_sum[blockIdx.x] = p_sum[blockIdx.x * blockDim.x] ;
			}
				
			for (unsigned int s = gridDim.x/2 ; s> 0 ; s >>= 1)
			{	
				if (index < s/2)
				{
					// summation of r*rT
					r_squared[index] = r_squared[index] + r_squared[index +  s] ;
					//summation of r*a*rT
					p_sum[index] = p_sum[index] +  p_sum[index +  s] ;
				}
				__syncthreads();
			}
		}
		
		float alpha = r_squared[0]/p_sum[0] ;
		x[index] = x[index] + alpha * local_r ;
	}
}

__global__ void cg_one(float* a , int * indeces , float* b , float* x,float * r ,float * r_squared ,float * p_sum ,int size) 
{
	int index = blockDim.x * blockIdx.x + threadIdx.x ;
	int local_index = threadIdx.x ;
	/*float local_a[3] = {a[3*index],a[3*index + 1],a[3*index + 2]} ;
	int local_indeces[3]  = {indeces[3*index],indeces[3*index + 1],indeces[3*index + 2]} ;*/
	__shared__ float shared_r_squared[1024] ;
	__shared__ float shared_p_sum[1024] ;

	shared_r_squared[index] = 0 ; // for extra numbers
	shared_p_sum[index] = 0 ;
	
	if (index < size)
	{
		float sum = 0 ;
		
		for (int i = 0 ; i<3 ; i++)
		{
			sum += a[3*index  + i] * x[indeces[3*index + i]] ;
		}
		
		float local_r = b[index] - sum ;	
		r[index] = local_r;


		for (int i = 0 ; i<3 ; i++)
		{
			shared_p_sum[index] += a[3*index  + i] * r[indeces[3*index + i]] ;
		}
		
		shared_r_squared[local_index] = local_r * local_r ;
		shared_p_sum[local_index] = shared_p_sum[index] * local_r ;
	}
	
	__syncthreads() ;

	for (unsigned int s = blockDim.x/2 ; s> 0 ; s >>= 1)
	{	
		if (threadIdx.x < s)
		{
			shared_r_squared[threadIdx.x] = shared_r_squared[threadIdx.x] + shared_r_squared[threadIdx.x +s] ;
			shared_p_sum[threadIdx.x] = shared_p_sum[threadIdx.x] + shared_p_sum[threadIdx.x +s] ;
			__syncthreads() ;
		}
			
	}	
	r_squared[blockIdx.x] = shared_r_squared[0] ;
	p_sum[blockIdx.x] = shared_p_sum[0] ;

}

__global__ void cg_two(float * r_squared ,float * p_sum ,int size) 
{
	int index = threadIdx.x ;
	__shared__ float shared_r_squared[1024] ;
	__shared__ float shared_p_sum[1024] ;

	if (index < size)
	{
		shared_r_squared[index] = r_squared[index]  ;
		shared_p_sum[index] = p_sum[index]  ;
	} else
	{
		shared_r_squared[index] = 0 ;
		shared_p_sum[index] = 0 ;
	}
	__syncthreads() ;


	for (unsigned int s = blockDim.x/2 ; s> 0 ; s >>= 1)
	{	
		if (threadIdx.x < s)
		{
			shared_r_squared[threadIdx.x] = shared_r_squared[threadIdx.x] + shared_r_squared[threadIdx.x +s] ;
			shared_p_sum[threadIdx.x] = shared_p_sum[threadIdx.x] + shared_p_sum[threadIdx.x +s] ;
			__syncthreads() ;
		}	
	}	
	if(threadIdx.x == 0)
	{
		//alpha
		r_squared[blockIdx.x] = shared_r_squared[0]/shared_p_sum[0] ;
	}
}

__global__ void cg_three(float * x ,float * r,float * r_squared ,int size) 
{
	int index = blockDim.x * blockIdx.x + threadIdx.x ;
	float alpha = r_squared[0] ;
	x[index] = x[index] + alpha * r[index] ;
}

void cg(const int size , char* file_name)
{
	//initialize our test cases

	float *values = (float *)malloc(3 * size * sizeof(float));
	int *indeces = (int *)malloc(3 * size * sizeof(int));
	float *x = (float *)malloc(size * sizeof(float));
	float *y = (float *)malloc(size * sizeof(float));
	float *output = (float *)malloc(size * sizeof(float));

	char* values_file_name = concat(file_name,"/basic/values.txt") ;
	char* indeces_file_name = concat(file_name,"/basic/indeces.txt");
	char* y_file_name = concat(file_name,"/right_hand_side.txt");
	char* output_file_name = concat(file_name,"/output.txt");

	FILE *values_file = fopen(values_file_name, "r");
	FILE *indeces_file = fopen(indeces_file_name, "r");
	FILE *y_file = fopen(y_file_name, "r");
	FILE *output_file = fopen(output_file_name, "r");

	for (int i = 0 ; i < size ; i++)
	{	
		fscanf(y_file, "%f", &y[i]);
		fscanf(output_file, "%f", &output[i]);
		x[i] = 0 ;
	}

	for (int i = 0 ; i< 3 * size ; i++)
	{
		fscanf(values_file, "%f", &values[i]);
		fscanf(indeces_file, "%d", &indeces[i]);	
	}
	
	float* dev_values = 0;
	int* dev_indeces = 0 ;
	float* dev_y = 0;
	float* dev_x = 0;
	float* dev_r = 0 ;
	float* dev_r_squared = 0 ;
	float* dev_p_sum = 0;

	int fraction = ceil(size/1024.0) ;
	int number_of_blocks = 100 ;

    hipSetDevice(0);
	
    // Allocate GPU buffers
    hipMalloc((void**)&dev_values, 3 * size * sizeof(float));
	hipMalloc((void**)&dev_indeces, 3 * size * sizeof(int));
    hipMalloc((void**)&dev_y, size * sizeof(float));
    hipMalloc((void**)&dev_x, size * sizeof(float));
	hipMalloc((void**)&dev_r, size * sizeof(float));
	hipMalloc((void**)&dev_r_squared, number_of_blocks * sizeof(float));
	hipMalloc((void**)&dev_p_sum, number_of_blocks * sizeof(float));
   
    // Copy input vectors from host memory to GPU buffers.
	hipMemcpyAsync(dev_values, values, 3 * size * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpyAsync(dev_indeces, indeces, 3 * size * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpyAsync(dev_y, y, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpyAsync(dev_x, x, size * sizeof(float), hipMemcpyHostToDevice);
	
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

    // Launch a kernel on the GPU with one thread for each row.
	cg_one<<<2,12>>>(dev_values , dev_indeces , dev_y ,  dev_x, dev_r , dev_r_squared , dev_p_sum , size) ;
	cg_two<<<1,2>>>(dev_r_squared ,dev_p_sum ,size);
	cg_three<<<2,12>>>( dev_x ,dev_r,dev_r_squared , size);
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
	hipDeviceSynchronize();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	printf ("Time for the kernel: %f ms\n", time);

    // Copy output vector from GPU buffer to host memory.
    hipMemcpy(x, dev_x, size * sizeof(float), hipMemcpyDeviceToHost);
	
	printf("%f\n",x[0]);
	printf("%f\n",x[1]);
	printf("%f\n",x[2]);
	printf("%f\n",x[size -2]);
	printf("%f\n",x[size -1]);
	hipDeviceReset();
	hipFree(dev_values);
	hipFree(dev_indeces) ;
	hipFree(dev_y);
	hipFree(dev_x);
	hipFree(dev_r) ;
	hipFree(dev_r_squared) ;
	hipFree(dev_p_sum) ;

	system("pause");
}

char* concat(char *s1, char *s2)
{
    char *result = (char *)malloc(strlen(s1)+strlen(s2)+1);
    strcpy(result, s1);
    strcat(result, s2);
    return result;
}

int main()
{
	cg(24,"C:/Users/youssef/Desktop/numerical-solutions-gpu/cg/cg/test_cases/24");
	return 1 ;
}