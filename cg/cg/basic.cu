#include "hip/hip_runtime.h"

#include <stdio.h>
#include<stdlib.h>
#include <string.h>
#include <math.h>

char* concat(char *s1, char *s2);

__global__  void cg_global(float* a , int * indeces , float* b , float* x,int size) 
{
	int index = blockDim.x * blockIdx.x + threadIdx.x ;
	__shared__  float r[1000] ;
	__shared__  float r_squared[1000] ;
	__shared__  float p_sum[1000] ;

	if (index < size)
	{
		float sum = 0 ;

		for (int i = 0 ; i<3 ; i++)
		{
			sum += a[i + 3*index] * x[indeces[i + 3*index]] ;
		}
		
		r[index] = b[index] - sum ;	
		p_sum[index] = 0 ;
		__syncthreads();

		for (int i = 0 ; i<3 ; i++)
		{
			p_sum[index] += a[i + 3*index] * r[indeces[i + 3*index]] ;
		}
		
		//calc alpha
		r_squared[index] = r[index] * r[index] ;
		p_sum[index] = p_sum[index] * r[index] ;
				
		for (unsigned int s = size/2 ; s> 0 ; s >>= 1)
		{
			
			if (index < size/2)
			{
				// summation of r*rT
				r_squared[index] = r_squared[index] + r_squared[index + s] ;
				//summation of r*a*rT
				p_sum[index] = p_sum[index] +  p_sum[index + s] ;
			}
			__syncthreads();
		}
		
		float alpha = r_squared[0]/p_sum[0] ;
		
		x[index] = x[index] + alpha * r[index] ;

	}

}


__global__ void cg_full_global(float* a , int * indeces , float* b , float* x,float * r ,float * r_squared ,float * p_sum ,int size) 
{
	int index = blockDim.x * blockIdx.x + threadIdx.x ;
	/*float local_a[3] = {a[3*index],a[3*index + 1],a[3*index + 2]} ;
	int local_indeces[3]  = {indeces[3*index],indeces[3*index + 1],indeces[3*index + 2]} ;*/
	
	if (index < size)
	{
		float sum = 0 ;
		
		for (int i = 0 ; i<3 ; i++)
		{
			sum += a[3*index  + i] * x[indeces[3*index + i]] ;
		}
		
		float local_r = b[index] - sum ;	
		r[index] = local_r;

		p_sum[index] = 0 ;
		__syncthreads();

		for (int i = 0 ; i<3 ; i++)
		{
			p_sum[index] += a[3*index  + i] * r[indeces[3*index + i]] ;
		}
		
		//calc alpha
		r_squared[index] = local_r * local_r ;
		p_sum[index] = p_sum[index] * local_r ;
		
		//sum inside block
		for (unsigned int s = blockDim.x/2 ; s> 0 ; s >>= 1)
		{	
			if (threadIdx.x < s/2)
			{
				// summation of r*rT
				r_squared[index] = r_squared[index] + r_squared[index + s] ;
				//summation of r*a*rT
				p_sum[index] = p_sum[index] +  p_sum[index + s] ;
			}
			__syncthreads();
		}
		
		// sum between blocks - 1024 or 2048 blocks only
		int max_index_of_needed_blocks = gridDim.x/1025 ;

		if (blockIdx.x <= max_index_of_needed_blocks && gridDim.x > 1)
		{
				r_squared[blockIdx.x] = r_squared[blockIdx.x * blockDim.x] ;
				p_sum[blockIdx.x] = p_sum[blockIdx.x * blockDim.x] ;
				__syncthreads();

			for (unsigned int s = gridDim.x/2 ; s> 0 ; s >>= 1)
			{	
				if (index < s/2)
				{
					// summation of r*rT
					r_squared[index] = r_squared[index] + r_squared[index +  s] ;
					//summation of r*a*rT
					p_sum[index] = p_sum[index] +  p_sum[index +  s] ;
				}
				__syncthreads();
			}
		}
		
		float alpha = r_squared[0]/p_sum[0] ;
		x[index] = x[index] + alpha * local_r ;
	}
}

void cg(const int size , char* file_name)
{
	//initialize our test cases

	float *values = (float *)malloc(3 * size * sizeof(float));
	int *indeces = (int *)malloc(3 * size * sizeof(int));
	float *x = (float *)malloc(size * sizeof(float));
	float *y = (float *)malloc(size * sizeof(float));
	float *output = (float *)malloc(size * sizeof(float));

	char* values_file_name = concat(file_name,"/basic/values.txt") ;
	char* indeces_file_name = concat(file_name,"/basic/indeces.txt");
	char* y_file_name = concat(file_name,"/right_hand_side.txt");
	char* output_file_name = concat(file_name,"/output.txt");

	FILE *values_file = fopen(values_file_name, "r");
	FILE *indeces_file = fopen(indeces_file_name, "r");
	FILE *y_file = fopen(y_file_name, "r");
	FILE *output_file = fopen(output_file_name, "r");

	for (int i = 0 ; i < size ; i++)
	{	
		fscanf(y_file, "%f", &y[i]);
		fscanf(output_file, "%f", &output[i]);
		x[i] = 0 ;
	}

	for (int i = 0 ; i< 3 * size ; i++)
	{
		fscanf(values_file, "%f", &values[i]);
		fscanf(indeces_file, "%d", &indeces[i]);	
	}
	
	float* dev_values = 0;
	int* dev_indeces = 0 ;
	float* dev_y = 0;
	float* dev_x = 0;
	float* dev_r = 0 ;
	float* dev_r_squared = 0 ;
	float* dev_p_sum = 0;

	int fraction = ceil(size/1024.0) ;

    hipSetDevice(0);
	
    // Allocate GPU buffers
    hipMalloc((void**)&dev_values, 3 * size * sizeof(float));
	hipMalloc((void**)&dev_indeces, 3 * size * sizeof(int));
    hipMalloc((void**)&dev_y, size * sizeof(float));
    hipMalloc((void**)&dev_x, size * sizeof(float));
	hipMalloc((void**)&dev_r, size * sizeof(float));
	hipMalloc((void**)&dev_r_squared, size * sizeof(float));
	hipMalloc((void**)&dev_p_sum, size * sizeof(float));
   
    // Copy input vectors from host memory to GPU buffers.
	hipMemcpyAsync(dev_values, values, 3 * size * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpyAsync(dev_indeces, indeces, 3 * size * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpyAsync(dev_y, y, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpyAsync(dev_x, x, size * sizeof(float), hipMemcpyHostToDevice);
	
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

    // Launch a kernel on the GPU with one thread for each row.
	//cg_local<<<2,900>>>(dev_values,dev_indeces,dev_y,dev_x,dev_r,dev_p_sum,size);
	//cg_global<<<1,420>>>(dev_values,dev_indeces,dev_y,dev_x,size);
	cg_full_global<<<504,200>>>(dev_values,dev_indeces,dev_y,dev_x,dev_r,dev_r_squared,dev_p_sum,size);
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
	hipDeviceSynchronize();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	printf ("Time for the kernel: %f ms\n", time);

    // Copy output vector from GPU buffer to host memory.
    hipMemcpyAsync(x, dev_x, size * sizeof(float), hipMemcpyDeviceToHost);
	
	printf("%f\n",x[0]);
	printf("%f\n",x[1]);
	printf("%f\n",x[2]);
	printf("%f\n",x[3]);
	printf("%f\n",x[83]);
	printf("%f\n",x[size -3]);
	printf("%f\n",x[size -2]);
	printf("%f\n",x[size -1]);
	hipDeviceReset();
	system("pause");
}

char* concat(char *s1, char *s2)
{
    char *result = (char *)malloc(strlen(s1)+strlen(s2)+1);
    strcpy(result, s1);
    strcat(result, s2);
    return result;
}

int main()
{
	cg(100800,"C:/Users/youssef/Desktop/numerical-solutions-gpu/cg/cg/test_cases/100800");
	return 1 ;
}